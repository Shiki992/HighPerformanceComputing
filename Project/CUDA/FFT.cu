#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <ctime>
#include <time.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <hipfft/hipfft.h>
#include <fstream>
#include <bits/stdc++.h>

using namespace std;
typedef float2 Complex;





__global__ void ComplexMUL(Complex *a, Complex *b)
{
    int i = threadIdx.x;
    a[i].x = a[i].x * b[i].x - a[i].y*b[i].y;
    a[i].y = a[i].x * b[i].y + a[i].y*b[i].x;
}


int main()
{


    int N = 8192;
    int SIZE = N*N;

    cout<<"Initalizing values for Matrix 1...\n";
    Complex *fg = new Complex[SIZE];
    for (int i = 0; i < SIZE; i++){
        fg[i].x = pow(2,15)+rand()+0.13246549884; 
        fg[i].y = pow(2,15)+rand()+0.13246549884;
    }
    cout<<"Initalizing values for Matrix 2...\n";
    Complex *fig = new Complex[SIZE];
    for (int i = 0; i < SIZE; i++){
        fig[i].x = pow(2,15)+rand()+0.13246549884; // 
        fig[i].y = pow(2,15)+rand()+0.13246549884;
    }
    cout<<"Initalized values...\nDONE!\n";
    
    // for (int i = 0; i < 24; i=i+5)
    // {
    //     cout << fg[i].x << " " << fg[i + 1].x << " " << fg[i + 2].x << " " << fg[i + 3].x << " " << fg[i + 4].x << endl;
    // }
    // cout << "----------------" << endl;
    // for (int i = 0; i < 24; i = i + 5)
    // {
    //     cout << fig[i].x << " " << fig[i + 1].x << " " << fig[i + 2].x << " " << fig[i + 3].x << " " << fig[i + 4].x << endl;
    // }
    cout << "----------------" << endl;

    int mem_size = sizeof(Complex)* SIZE;


    clock_t start, end;

    
    hipfftComplex *d_signal;
    hipMalloc((void **) &d_signal, mem_size); 
    hipMemcpy(d_signal, fg, mem_size, hipMemcpyHostToDevice);

    
    hipfftComplex *d_filter_kernel;
    hipMalloc((void **)&d_filter_kernel, mem_size);
    hipMemcpy(d_filter_kernel, fig, mem_size, hipMemcpyHostToDevice);
    
  hipfftHandle plan;
    hipfftPlan2d(&plan, N, N, HIPFFT_C2C);
     
    start=clock();
    // Transform signal and filter
    printf("Transforming signal hipfftExecR2C\n");
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD);
    hipfftExecC2C(plan, (hipfftComplex *)d_filter_kernel, (hipfftComplex *)d_filter_kernel, HIPFFT_FORWARD);

    printf("Launching Complex multiplication<<< >>>\n");
    ComplexMUL <<< 512, 512 >> >(d_signal, d_filter_kernel);

    // Transform signal back
    printf("Transforming signal back hipfftExecC2C\n");
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_BACKWARD);

    Complex *result = new Complex[SIZE];
    hipMemcpy(result, d_signal, sizeof(Complex)*SIZE, hipMemcpyDeviceToHost);
    end=clock(); 
  
    double time = double(end - start)/(CLOCKS_PER_SEC/double(1000.0)); 

    

    // for (int i = 0; i < SIZE; i=i+5)
    // {
    //     cout << result[i].x << " " << result[i + 1].x << " " << result[i + 2].x << " " << result[i + 3].x << " " << result[i + 4].x << endl;
    // }
    cout<<"Time Taken: "<<fixed<<time<<setprecision(5)<<" ms"<<endl;

    delete result, fg, fig;
    hipfftDestroy(plan);
    //hipfftDestroy(plan2);
    hipFree(d_signal);
    hipFree(d_filter_kernel);

}
